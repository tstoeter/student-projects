#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <list>
#include <vector>

#include "cutil.h"

#include "grains.h"
#include "visual.h"
#include "bmp.h"

using namespace std;


// simulation parameters
unsigned int w, h, d;
float kT = 1.0f;
int *rands;

// statistics variables
unsigned int *domain, *stats, ngrains;
list<int> grain_list;

// opengl buffers
struct hipGraphicsResource *pbores;
GLuint pbo, texid;

texture<unsigned int, 2, hipReadModeElementType> tex;
texture<unsigned int, 3, hipReadModeElementType> tex3d;
hipArray* d_array;

/////// mcp simulation code ////////////////////////////////

// relative nearest neighbor indices
__constant__ int nn2d[8*2] = {1,0, -1,0, 0,1, 0,-1, 1,1, -1,1, -1,-1, 1,-1};
__constant__ int nn3d[26*3] = {1,0,0, -1,0,0, 0,1,0, 0,-1,0, 1,1,0, -1,1,0, -1,-1,0, 1,-1,0,
			0,0,1, 1,0,1, -1,0,1, 0,1,1, 0,-1,1, 1,1,1, -1,1,1, -1,-1,1, 1,-1,1,
			0,0,-1, 1,0,-1, -1,0,-1, 0,1,-1, 0,-1,-1, 1,1,-1, -1,1,-1, -1,-1,-1, 1,-1,-1};

// delta function
__device__ int delta(int qi, int qj)
{
	return (qi == qj);
}

// convert absolute texture coords to (0,1) - normalized texture coords
__device__ float abs2norm(float x, float w)
{
	return (x+0.5)/w;
}

// computes energy difference of a cell (x,y) based on a new orientation qn and nearest neighbor orientations
__device__ int energy_diff2d_tex(int x, int y, int w, int h, int qo, int qn)
{
	int j, e=0, qj;

	for (j=0; j<16; j+=2)
	{
		qj = tex2D(tex, abs2norm(x+nn2d[j+0],w), abs2norm(y+nn2d[j+1],h));
		e += delta(qj, qo) - delta(qj, qn);
	}

	return e;
}

// computes energy difference of a cell (x,y) based on a new orientation qn and nearest neighbor orientations
__device__ int energy_diff3d_tex(int x, int y, int z, int w, int h, int d, int qo, int qn)
{
	int j, e=0, qj;

	for (j=0; j<78; j+=3)
	{
		qj = tex3D(tex3d, abs2norm(x+nn3d[j+0],w), abs2norm(y+nn3d[j+1],h), abs2norm(z+nn3d[j+2],d));
		e += delta(qj, qo) - delta(qj, qn);
	}

	return e;
}

// re-orientation probability depending on energy difference
__device__ float prob(int e, float kT)
{
	return __expf(-e/kT);
}

// 16-bit linear congruentional generator for pseudo random numbers
__device__ int lcg_rand(int x)
{
	return (__umul24(19961, x) + 13) % 0xFFFF;
}

// do one monte-carlo-potts iteration
__global__ void mcs2d(int *domain, int w, int h, float kT, int *rands)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int i = __umul24(y,w) + x;

	int rand1 = rands[i];
	int rand2 = lcg_rand(rand1);

	// store prng state
	rands[i] = lcg_rand(rand2);

	int j = (rand1 & 7) << 1;
	int xx = x + nn2d[j+0];
	int yy = y + nn2d[j+1];

	int qo = tex2D(tex, abs2norm(x,w), abs2norm(y,h));
	int qn = tex2D(tex, abs2norm(xx,w), abs2norm(yy,h));

	if (qo != qn)
	{
		int e = energy_diff2d_tex(x, y, w, h, qo, qn);

		// probabilistic re-orientation
		if ((float)rand2/0xFFFF < prob(e,kT))
			domain[i] = qn;
	}
}

// kernel for bitmaps with w > 512
__global__ void mcs2d512(int *domain, int w, int h, float kT, int *rands)
{
	int i = (blockIdx.x << 9) + threadIdx.x;
	int x = i % w;
	int y = i / w;

	int rand1 = rands[i];
	int rand2 = lcg_rand(rand1);

	// store prng state
	rands[i] = lcg_rand(rand2);

	int j = (rand1 & 7) << 1;
	int xx = x + nn2d[j+0];
	int yy = y + nn2d[j+1];

	int qo = tex2D(tex, abs2norm(x,w), abs2norm(y,h));
	int qn = tex2D(tex, abs2norm(xx,w), abs2norm(yy,h));

	if (qo != qn)
	{
		int e = energy_diff2d_tex(x, y, w, h, qo, qn);

		// probabilistic re-orientation
		if ((float)rand2/0xFFFF < prob(e,kT))
			domain[i] = qn;
	}
}


// do one monte-carlo-potts iteration in 3d
__global__ void mcs3d(int *domain, int w, int h, int d, float kT, int *rands)
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int z = blockIdx.y;
	int i = __umul24(z, __umul24(w,h)) + __umul24(y,w) + x;

	int rand1 = rands[i];
	int rand2 = lcg_rand(rand1);

	// store prng state
	rands[i] = lcg_rand(rand2);

	int j = __umul24(3, (rand1 % 26));
	int xx = x + nn3d[j+0];
	int yy = y + nn3d[j+1];
	int zz = z + nn3d[j+2];

	int qo = tex3D(tex3d, abs2norm(x,w), abs2norm(y,h), abs2norm(z,d));
	int qn = tex3D(tex3d, abs2norm(xx,w), abs2norm(yy,h), abs2norm(zz,d));

	if (qo != qn)
	{
		int e = energy_diff3d_tex(x, y, z, w, h, d, qo, qn);

		// probabilistic re-orientation
		if ((float)rand2/0xFFFF < prob(e,kT))
			domain[i] = qn;
	}
}

////////////////////////////////////////////////////////////////////

void run_mcs2d(int w, int h, float kT)
{
	int *d_out;
	hipGraphicsMapResources(1, &pbores, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void**)&d_out, &num_bytes, pbores);

	// run cuda kernel
	if (w <= 512)
		mcs2d<<<h,w>>>(d_out, w, h, kT, rands);
	else
		mcs2d512<<<(w*h)/512,512>>>(d_out, w, h, kT, rands);

	CUT_CHECK_ERROR("kernel launch failure");

//	hipDeviceSynchronize();

	// copy output data to texture for next iteration
	hipUnbindTexture(tex);
	hipMemcpyToArray( d_array, 0, 0, d_out, w * h * sizeof(int), hipMemcpyDeviceToDevice);
	hipBindTextureToArray(tex, d_array);

	hipGraphicsUnmapResources(1, &pbores, 0);
}

void run_mcs3d(int w, int h, int d, float kT)
{
	int *d_out;
	hipGraphicsMapResources(1, &pbores, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void**)&d_out, &num_bytes, pbores);

	dim3 gridDim(h,d);

	mcs3d<<<gridDim,w>>>(d_out, w, h, d, kT, rands);

	CUT_CHECK_ERROR("kernel launch failure");

//	hipDeviceSynchronize();

	// copy output data to texture for next iteration
	hipUnbindTexture(tex);
	hipMemcpy3DParms params = {0};
	params.srcPtr = make_hipPitchedPtr(d_out, 0, w, h);
	params.extent = make_hipExtent(w,h,d);
	params.dstArray = d_array;
	params.kind = hipMemcpyDeviceToDevice;
	hipMemcpy3D(&params);
	CUT_CHECK_ERROR("resource allocation failure");
	hipBindTextureToArray(tex, d_array);

	hipGraphicsUnmapResources(1, &pbores, 0);
}

void init_stats()
{
	// zero stats array
	memset(stats, 0, 0x00ffffff*sizeof(int));
	ngrains = 0;

	// and count occurences
	for (int i=0; i<w*h*d; i++)
		stats[domain[i]>>8]++;

	// create grain list
	for (int i=0; i<0x00ffffff; i++)
		if (stats[i] > 0)
			grain_list.push_back(i);
}

void update_stats()
{
	// zero stats array
	memset(stats, 0, 0x00ffffff*sizeof(int));
	ngrains = 0;

	// and count occurences
	for (int i=0; i<w*h*d; i++)
		stats[domain[i]>>8]++;

	for (list<int>::iterator it=grain_list.begin(); it!=grain_list.end();)
		if (stats[*it] == 0)
			it = grain_list.erase(it);
		else
			it++;
}

void print_stats(int step)
{
	printf("step=%d\t #grains=%d\n", step, grain_list.size());
}

void print_grains(int step)
{
	printf("%d", step);
	for (list<int>::iterator it=grain_list.begin(); it!=grain_list.end(); it++)
		printf(", %X, %d", *it, stats[*it]);

	printf("\n");
}

void init_rands(unsigned long n, int *rands)
{
	int i;
	vector<int> tmp(n);

	printf("creating random numbers\n");

	srand(time(NULL));

	for (i=0; i<n; i++)
		tmp[i] = rand() % RAND_MAX;

	printf("uploading to gpu\n");

	hipMemcpy((void **)rands, (void **)(&tmp[0]), n*sizeof(int), hipMemcpyHostToDevice);
	CUT_CHECK_ERROR("resource allocation failure");
}

int init_grains(const char *filename)
{
	unsigned int *data;
	int dims = 2;

	if (filename == NULL)
	{
		w = h = 256; d = 1;

		data = (unsigned int *)malloc(w*h*sizeof(unsigned int));

		for (int i=0; i<w*h; i++)
			data[i] = (rand() & 0x00ffffff) << 8;
	}
	else if (strcmp(&filename[strlen(filename)-4], ".bmp") == 0)
	{
		bmp_header head;
		bmp_info info;

		char *data24 = loadbmp(filename, &head, &info);

		w = info.width;
		h = info.height;
		d = 1;

		if (data24 == NULL || info.bpp != 24)
		{
			free(data24);
			printf("bad bitmap file\n");
			exit(1);
		}

		// convert to 32bit data
		data = (unsigned int *)malloc(w*h*sizeof(unsigned int));
		for (int i=0; i<w*h; i++)
			data[i] = (((int)data24[3*i+2] & 0x000000ff) << 24) + (((int)data24[3*i+1] & 0x000000ff) << 16) + (((int)data24[3*i+0] & 0x000000ff) << 8);
	}
	else if (strcmp(&filename[strlen(filename)-4], ".bm3") == 0)
	{
		dims = 3;

		bm3_info info;

		char *data24 = loadbm3(filename, &info);

		w = info.width;
		h = info.height;
		d = info.depth;

		if (data24 == NULL || info.bpp != 24)
		{
			free(data24);
			printf("bad bitmap file\n");
			exit(1);
		}

		// convert to 32bit data
		data = (unsigned int *)malloc(w*h*d*sizeof(unsigned int));
		for (int i=0; i<w*h*d; i++)
			data[i] = (((int)data24[3*i+0] & 0x000000ff) << 16) + (((int)data24[3*i+1] & 0x000000ff) << 8) + (((int)data24[3*i+2] & 0x000000ff) << 0) + 0xff000000;

	}

	printf("data converted\n");

	cudaGLSetGLDevice(0);
	CUT_CHECK_ERROR("device setting failure");

	// allocate opengl buffer for computing and upload data
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_PACK_BUFFER_ARB, pbo);
	glBufferData(GL_PIXEL_PACK_BUFFER_ARB, w*h*d*sizeof(unsigned int), data, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_PIXEL_PACK_BUFFER_ARB, 0);

	hipGraphicsGLRegisterBuffer(&pbores, pbo, cudaGraphicsMapFlagsNone);
	CUT_CHECK_ERROR("resource registration failure");

	printf("gpu memory allocated\n");

	if (dims == 2)
	{
		width = w;
		height = h;

		// copy image data to array
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
		hipMallocArray( &d_array, &channelDesc, w, h);
		hipMemcpyToArray(d_array, 0, 0, data, w*h*sizeof(int), hipMemcpyHostToDevice);

		// set texture parameters
		tex.addressMode[0] = hipAddressModeWrap;
		tex.addressMode[1] = hipAddressModeWrap;
		tex.filterMode = hipFilterModePoint;
		// tex coords need to be normalized in cuda 3.2 for texture wrapping to work
		tex.normalized = true;

		// Bind the array to the texture
		hipBindTextureToArray(tex, d_array, channelDesc);

		// create 2d textures for display
		glGenTextures(1, &texid);
		glBindTexture(GL_TEXTURE_2D, texid);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, w, h, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
		glBindTexture(GL_TEXTURE_2D, 0);
	}
	else if (dims == 3)
	{
		width = 512;
		height = 512;

		// copy image data to array
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
		hipMalloc3DArray(&d_array, &channelDesc, make_hipExtent(w,h,d));

		hipMemcpy3DParms params = {0};
		params.srcPtr = make_hipPitchedPtr(data, 4*w, w, h);
		params.extent = make_hipExtent(w,h,d);
		params.dstArray = d_array;
		params.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&params);
		CUT_CHECK_ERROR("resource allocation failure");

		// set texture parameters
		tex3d.addressMode[0] = hipAddressModeWrap;
		tex3d.addressMode[1] = hipAddressModeWrap;
		tex3d.addressMode[2] = hipAddressModeWrap;
		tex3d.filterMode = hipFilterModePoint;
		// tex coords need to be normalized in cuda 3.2 for texture wrapping to work
		tex3d.normalized = true;

		// Bind the array to the texture
		hipBindTextureToArray(tex3d, d_array, channelDesc);

		// create 3d textures for display
		glGenTextures(1, &texid);
		glBindTexture(GL_TEXTURE_3D, texid);
		// clamp to border so we get a black border around the volume and no parts of the volume repeated
		glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
		glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);
		glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_BORDER);
		glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
		glTexParameteri(GL_TEXTURE_3D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
		// enable the border

		// make texture
		glTexImage3D(GL_TEXTURE_3D, 0, GL_RGBA, w, h, d, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
		glBindTexture(GL_TEXTURE_3D, 0);

		// configure texture coordinate generation
		// for view-aligned slices
		float plane_x[] = {1.0f, 0.0f, 0.0f, 0.0f};
		float plane_y[] = {0.0f, 1.0f, 0.0f, 0.0f};

		glTexGenfv(GL_S, GL_OBJECT_PLANE, plane_x);
		glTexGenfv(GL_T, GL_OBJECT_PLANE, plane_y);

		glEnable(GL_TEXTURE_GEN_S);
		glEnable(GL_TEXTURE_GEN_T);
		glEnable(GL_TEXTURE_GEN_R);
	}

	free(data);

	printf("data uploaded to gpu. %d %d %d\n", w, h, d);

	// init random number generator
	hipMalloc((void**)&rands, w*h*d*sizeof(int));
	printf("gpu mem for rands allocated\n");
	init_rands(w*h*d, rands);
	printf("rands created and transfered\n");

	// allocate download buffer
	domain = new unsigned int[w*h*d];
	stats = new unsigned int[0x00ffffff];

	printf("stats buffers allocated\n");

	// resize window
	glutReshapeWindow(width, height);

	return dims;
}

void download_grains()
{
	int *dptr;

	hipGraphicsMapResources(1, &pbores, 0);
	size_t num_bytes; 
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, pbores);

	hipMemcpy((void **)domain, (void **)dptr, num_bytes, hipMemcpyDeviceToHost);
	
	hipGraphicsUnmapResources(1, &pbores, 0);
}

void store_grains(const char *filename)
{
	if (filename == NULL)
		return;

	char *data24 = (char *)malloc(w*h*d*3);

	if (dims == 2)
	{
		// convert to 24bit data
		for (int i=0; i<w*h*d; i++)
		{
			data24[3*i+2] = (domain[i] >> 24) & 0xff;
			data24[3*i+1] = (domain[i] >> 16) & 0xff;
			data24[3*i+0] = (domain[i] >> 8) & 0xff;
		}		

		bmp_header head;
		bmp_info info;
		
		head.filesize = 2 + sizeof(bmp_header) + sizeof(bmp_info) + 3*w*h*d;
		head.dataoffset = 2 + sizeof(bmp_header) + sizeof(bmp_info);

		info.headersize = sizeof(bmp_info);
		info.numplanes = 1;
		info.width = w;
		info.height = h;
		info.bpp = 24;
		info.compression = 0;
		info.datasize = w*h*3;
		info.widthppm = info.heightppm = 3780;
		info.numcolors = info.impcolors = 0;

		savebmp(filename, &head, &info, data24);
	}
	else if (dims == 3)
	{
		// convert to 24bit data
		for (int i=0; i<w*h*d; i++)
		{
			data24[3*i+0] = (domain[i] >> 16) & 0xff;
			data24[3*i+1] = (domain[i] >> 8) & 0xff;
			data24[3*i+2] = (domain[i] >> 0) & 0xff;
		}		

		bm3_info info;

		info.width = w;
		info.height = h;
		info.depth = d;
		info.bpp = 24;
		info.format = 0;

		savebm3(filename, &info, data24);
	}

	free(data24);
}

void destroy_grains()
{
	hipFree(rands);
	free(domain);

	delete[] stats;

	if (dims == 2)
		hipUnbindTexture(tex);
	else
		hipUnbindTexture(tex3d);

	hipFreeArray(d_array);

	hipGraphicsUnregisterResource(pbores);

	glDeleteBuffersARB(1, &pbo);
	glDeleteTextures(1, &texid);
}

